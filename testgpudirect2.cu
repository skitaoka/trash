// Peer 間のメモリ転送速度を測る。
//
// nvcc -O2 -arch=sm_35 –Xcompiler "–fopenmp" testgpudirect2.cu
//

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <omp.h>

void _check(hipError_t const e)
{
  if (e) {
    std::fprintf(stderr, "%s\n", ::hipGetErrorString(e));
    std::exit(1);
  }
}

int main(int argc, char * argv[])
{
  if (argc < 4) {
    std::fprintf(stderr, "Usage: %s gpu_from gpu_to bytes\n", argv[0]);
    return 1;
  }

  int const gpu_from = std::atoi(argv[1]);
  int const gpu_to   = std::atoi(argv[2]);
  int const size     = std::atoi(argv[3]);

  int canAccessPeer;
  _check(::hipDeviceCanAccessPeer(&canAccessPeer, gpu_to, gpu_from));
  if (!canAccessPeer) {
    std::fprintf(stderr, "(%d-%d) cannot access peer.\n", gpu_from, gpu_to);
    return 1;
  }
  //_check(::cudaDeviceEnablePeerAccess(gpu_to, gpu_from));

  char * data0 = NULL; // GPU 0 上のメモリ
  char * data1 = NULL; // GPU 1 上のメモリ
  char * data2 = NULL; // CPU 上のメモリ

  _check(::hipSetDevice(gpu_from));
  _check(::hipMalloc(&data0, size));

  _check(::hipSetDevice(gpu_to));
  _check(::hipMalloc(&data1, size));

  _check(::hipHostAlloc(&data2, size, hipHostMallocPortable));

  _check(::hipDeviceSynchronize());

  // GPU Direct による転送
  {
    double       const wstart = ::omp_get_wtime();
    std::clock_t const cstart = std::clock();
    {
      _check(::hipMemcpyPeer(data1, gpu_to, data0, gpu_from, size));
      _check(::hipDeviceSynchronize());
    }
    std::clock_t const cend = std::clock();
    double       const wend = ::omp_get_wtime();

    double const wtime = (wend - wstart);
    double const ctime = (cend - cstart) / double(CLOCKS_PER_SEC);
    std::fprintf(stderr, "D wtime = %lf sec, %lf GB/sec\n", wtime, size / 1000000000.0 / wtime);
    std::fprintf(stderr, "D ctime = %lf sec, %lf GB/sec\n", ctime, size / 1000000000.0 / ctime);
  }

  // CPU を介した転送
  {
    double       const wstart = ::omp_get_wtime();
    std::clock_t const cstart = std::clock();
    {
      _check(::hipSetDevice(gpu_from));
      _check(::hipMemcpyAsync(data2, data0, size, hipMemcpyDeviceToHost));
      _check(::hipDeviceSynchronize());
      _check(::hipSetDevice(gpu_to));
      _check(::hipMemcpyAsync(data1, data2, size, hipMemcpyHostToDevice));
      _check(::hipDeviceSynchronize());
    }
    std::clock_t const cend = std::clock();
    double       const wend = ::omp_get_wtime();

    double const wtime = (wend - wstart);
    double const ctime = (cend - cstart) / double(CLOCKS_PER_SEC);
    std::fprintf(stderr, "I wtime = %lf sec, %lf GB/sec\n", wtime, size / 1000000000.0 / wtime);
    std::fprintf(stderr, "I ctime = %lf sec, %lf GB/sec\n", ctime, size / 1000000000.0 / ctime);
  }

  _check(::hipSetDevice(gpu_from));
  _check(::hipFree(data0));
  _check(::hipDeviceReset());

  _check(::hipSetDevice(gpu_to));
  _check(::hipFree(data1));
  _check(::hipHostFree(data2));
  _check(::hipDeviceReset());

  return 0;
}

