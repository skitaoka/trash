// __ballot により warp 中のスレッドを同期化するサンプル。
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void
  kernel(int volatile * const a)
{
  int const tid = threadIdx.x & 31;
  int const i = threadIdx.x + blockIdx.x * blockDim.x;
  for (int j = 0; j < 32; ++j) {
    int const b = __ballot(1);
    if ((b & ((1U << tid) - 1U)) == 0) {
      a[i] = b;
      break;
    }
  }
}
 
int main(int argc, char* argv[])
{
  ::hipSetDevice(0);
  {
    static int const n = 32;
    int a[n] = {};

    int * b;
    ::hipMalloc(&b, sizeof(int) * n);
    ::hipMemcpyAsync(b, a, sizeof(int) * n, hipMemcpyHostToDevice);
    kernel<<<1, n>>>(b);
    ::hipMemcpy(a, b, sizeof(int) * n, hipMemcpyDeviceToHost);
    ::hipFree(b);

    for (int i = 0; i < n; ++i) {
      unsigned const v = a[i];
      for (int k = 0; k < 32; ++k) {
        std::cout << ((v >> (31-k))&1);
      }
      std::cout << std::endl;
    }
  }
  ::hipDeviceReset();

  return 0;
}
